#include "hip/hip_runtime.h"
#include <stdio.h>
#include <algorithm>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>

#define CSC(call)																							\
do {																										\
	hipError_t status = call;																				\
	if (status != hipSuccess) {																			\
		fprintf(stderr, "ERROR is %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status));	\
		exit(0);																							\
	}																										\
} while(0)	

struct comparator {												
	__host__ __device__ bool operator()(double a, double b) {
		return fabs(a) < fabs(b);
	}
};

__global__ void swap_kernel(double *sub_matrix, int n, int m, int y, int x, int max_row_index){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (y+idx < m){
        double tmp = sub_matrix[(y+idx)*n+x];
        sub_matrix[(y+idx)*n+x] = sub_matrix[(y+idx)*n+max_row_index];
        sub_matrix[(y+idx)*n+max_row_index] = tmp;
        idx += gridDim.x * blockDim.x;
    }
}

__global__ void kernel(double* sub_matrix, int n, int m, int x, int y) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    while (y+1+idy < m){
        idx = blockDim.x * blockIdx.x + threadIdx.x;
        while (x+1+idx < n){
            sub_matrix[(idy+y+1)*n+(idx+x+1)] -= sub_matrix[(idy+y+1)*n+x] * sub_matrix[y*n+(idx+x+1)]/sub_matrix[y*n+x];
            idx += blockDim.x * gridDim.x;
        }
        idy += blockDim.y * gridDim.y;
    }
}

int main() {
    comparator comp;
    int n, m;
    scanf("%d%d", &n, &m);
    double* matrix = (double*)malloc(sizeof(double)*n*m);
    for (int i = 0; i < n; ++i){
        for (int j = 0; j < m; ++j)
            scanf("%lf", &matrix[j*n+i]);
    }

    double *dev_matrix;
	CSC(hipMalloc(&dev_matrix, sizeof(double)*n*m));
	CSC(hipMemcpy(dev_matrix, matrix, sizeof(double)*n*m, hipMemcpyHostToDevice));

    int j = 0;
    for (int i = 0; i < m; ++i){
        thrust::device_ptr<double> p_matrix = thrust::device_pointer_cast(dev_matrix);
	    thrust::device_ptr<double> p_max_element = thrust::max_element(p_matrix+i*n+j, p_matrix+i*n+n, comp);
        if (fabs(*p_max_element) > 1e-7){
            if (j == n-1) {
                ++j;
                break;
            }
            int max_row_index = (int)(p_max_element-(p_matrix+i*n));
            if (j != max_row_index){
                swap_kernel<<<32, 32>>> (dev_matrix, n, m, i, j, max_row_index);
                CSC(hipGetLastError());
            }
            kernel<<< dim3(32, 32), dim3(32, 32) >>> (dev_matrix, n, m, j, i);
            CSC(hipGetLastError());
            ++j;
        }   
    }

    printf("%d", j);

    hipFree(dev_matrix);
	free(matrix);

	return 0;
}