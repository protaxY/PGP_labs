#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define CSC(call)																							\
do {																										\
	hipError_t status = call;																				\
	if (status != hipSuccess) {																			\
		fprintf(stderr, "ERROR is %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status));	\
		exit(0);																							\
	}																										\
} while(0)																									\

__global__ void kernel(long long n, double* a, double* b, double* c) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	while (idx < n) {
		c[idx] = a[idx] + b[idx];
		idx += gridDim.x * blockDim.x;
	}
}

int main () {
	long long n;
	scanf("%lld", &n);

	double* a = (double*)malloc(sizeof(double) * n);
	double* b = (double*)malloc(sizeof(double) * n);
	double* c = (double*)malloc(sizeof(double) * n);

	for (long long i = 0; i < n; ++i) {
		scanf("%lf", &a[i]);
	}
	for (long long i = 0; i < n; ++i) {
		scanf("%lf", &b[i]);
	}

	double* dev_a;
	double* dev_b;
	double* dev_c;

	CSC(hipMalloc(&dev_a, sizeof(double) * n));
	CSC(hipMalloc(&dev_b, sizeof(double) * n));
	CSC(hipMalloc(&dev_c, sizeof(double) * n));

	CSC(hipMemcpy(dev_a, a, sizeof(double) * n, hipMemcpyHostToDevice));
	CSC(hipMemcpy(dev_b, b, sizeof(double) * n, hipMemcpyHostToDevice));

	kernel<<<256, 256>>> (n, dev_a, dev_b, dev_c);
	CSC(hipGetLastError());

	CSC(hipMemcpy(c, dev_c, sizeof(double) * n, hipMemcpyDeviceToHost));

	for (long long i = 0; i < n; ++i) {
		printf("%.10e ", c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	free(a);

	free(b);
	free(c);

	return 0;
}